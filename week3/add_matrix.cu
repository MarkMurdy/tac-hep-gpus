
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


const int DSIZE_X = 256;
const int DSIZE_Y = 256;

__global__ void add_matrix(const float *A, const float *B, float *C, int size_x, int size_y)
{
    //FIXME:
    // Express in terms of threads and blocks
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int idy = blockDim.y*blockIdx.y + threadIdx.y;
    // Add the two matrices - make sure you are not out of range
    if (idx <  size_x && idy < size_y )
        C[idx*size_y + idy] = A[idx*size_y + idy] + B[idx*size_y + idy];

}

void print_matrix(const float *mat, int y_size, int x_max, int y_max)
{
    for (int i = 0; i < x_max; i++) {
        for (int j = 0; j < y_max; j++) {
            std::cout << mat[i*y_size + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main()
{

    // Create and allocate memory for host and device pointers 
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE_X*DSIZE_Y];
    h_B = new float[DSIZE_X*DSIZE_Y];
    h_C = new float[DSIZE_X*DSIZE_Y];
    int matrix_size = sizeof(float)*DSIZE_X*DSIZE_Y;
    hipMalloc(&d_A, matrix_size);
    hipMalloc(&d_B, matrix_size);
    hipMalloc(&d_C, matrix_size);

    // Fill in the matrices
    // FIXME
    for (int i = 0; i < DSIZE_X; i++) {
        for (int j = 0; j < DSIZE_Y; j++) {
            h_A[i*DSIZE_Y + j] = rand()/(float)RAND_MAX;
            h_B[i*DSIZE_Y + j] = rand()/(float)RAND_MAX;
            h_C[i*DSIZE_Y +j] = 0;
        }
    }

    // Copy from host to device
    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, matrix_size, hipMemcpyHostToDevice);

    // Launch the kernel
    // dim3 is a built in CUDA type that allows you to define the block 
    // size and grid size in more than 1 dimentions
    // Syntax : dim3(Nx,Ny,Nz)
    dim3 blockSize(16, 16);
    dim3 gridSize(16, 16);
    
    add_matrix<<<gridSize, blockSize>>>(d_A, d_B, d_C, DSIZE_X, DSIZE_Y);

    // Copy back to host 
    hipMemcpy(h_A, d_A, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);

    // Print and check some elements to make the addition was succesfull
    print_matrix(h_A, DSIZE_Y, 2, 5);
    print_matrix(h_B, DSIZE_Y, 2, 5);
    print_matrix(h_C, DSIZE_Y, 2, 5);

    // Free the memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}