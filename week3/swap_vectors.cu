
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_swap(float *A, float *B, float *C, int vsize) {

    //FIXME:
    // Express the vector index in terms of threads and blocks
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    // Swap the vector elements - make sure you are not out of range
    if (idx < vsize) {
        C[idx] = A[idx];
        A[idx] = B[idx];
        B[idx] = C[idx];
    }
}


int main() {


    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }
    for (int i = 0; i < 10; i++) {
        std::cout << h_A[i] << " ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_B[i] << " ";
    }
    std::cout << std::endl;


    // Allocate memory for host and device pointers 
    int vector_size = DSIZE*sizeof(float);
    hipMalloc(&d_A, vector_size);
    hipMalloc(&d_B, vector_size);
    hipMalloc(&d_C, vector_size);

    // Copy from host to device
    hipMemcpy(d_A, h_A, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, vector_size, hipMemcpyHostToDevice);

    // Launch the kernel
    vector_swap<<<grid_size, block_size>>>(d_A, d_B, d_C, DSIZE);

    // Copy back to host 
    hipMemcpy(h_A, d_A, vector_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, vector_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, vector_size, hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    for (int i = 0; i < 10; i++) {
        std::cout << h_A[i] << " ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_B[i] << " ";
    }
    std::cout << std::endl;

    // Free the memory 
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
